#include "hip/hip_runtime.h"


#include "Grid.cuh"
#include "VoxelsCA.cuh"
#include "BasePlate.cuh"
#include "TempField.cuh"
#include "iostream"
#include "vector"
#include <math.h>
#include <chrono>
#include <thread>
#include <algorithm>
#include "fstream"



static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HandleError( err ) (HandleError( err, __FILE__, __LINE__ ))


int main(int argc, char *argv[])
{
  //-----------------------------------------------
  // initialize and create base plate
  // set up all pointers for arrays in class
  double *d_lcoor,*d_lcoor2;
  // voxels
  int *d_gID,*d_ineighID,*d_neighptr,*d_vState,*d_itmp,nBlocks,
    nThreads,*d_ispvec;
  double *d_cTheta,*d_extents,*d_centroidOct;
  // tempfield
  double *d_Tempvals;
  // initialize class variables
  auto texec1 = std::chrono::high_resolution_clock::now();
  std::string filbaseOut,filout,filLogOut,filParamIn;
  filParamIn = argv[1];
  Grid g(filParamIn);
  Grid *d_g;
  int Ntot=g.nX[0]*g.nX[1]*g.nX[2];
  HandleError(hipMallocManaged((void**)&d_g,sizeof(Grid)));
  HandleError(hipMemcpy(d_g, &g, sizeof(Grid), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_lcoor,2*g.NpT*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_lcoor2,2*g.NpT*sizeof(double)));
  HandleError(hipMemcpy(d_lcoor,(g.lcoor), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  HandleError(hipMemcpy(d_lcoor2,(g.lcoor2), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  TempField TempF(g);
  TempF.InitializeAnalytic(g);
  TempField *d_TempF;
  HandleError(hipMallocManaged((void**)&d_TempF,sizeof(Grid)));
  HandleError(hipMalloc((void**)&d_Tempvals,Ntot*sizeof(double)));
  HandleError(hipMemcpy(d_TempF, &TempF, sizeof(TempField), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_ispvec,g.NpT*sizeof(int)));
  HandleError(hipMemcpy(d_ispvec,(TempF.ispvec), g.NpT*sizeof(int), hipMemcpyHostToDevice));
  std::vector<double> Sites;
  GenerateGrainSites(g,Sites);
  VoxelsCA vox(g);
  vox.nGrain = Sites.size()/3;
  double *d_Sites;
  HandleError(hipMallocManaged((void**)&d_Sites,vox.nGrain*3*sizeof(double)));
  HandleError(hipMemcpy(d_Sites,Sites.data(), vox.nGrain*3*sizeof(double), hipMemcpyHostToDevice));
  vox.cTheta=(double*)malloc(vox.nGrain*4*sizeof(double));
  VoxelsCA *d_vox;
  HandleError(hipMallocManaged((void**)&d_vox,sizeof(VoxelsCA)));
  HandleError(hipMemcpy(d_vox, &vox, sizeof(VoxelsCA), hipMemcpyHostToDevice));
  nThreads=1024;
  HandleError(hipMallocManaged((void**)&d_gID,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_vState,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_extents,Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_centroidOct,3*Ntot*sizeof(double)));
  HandleError(hipMemset(d_gID,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_vState,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_extents,0,Ntot*sizeof(double)));
  HandleError(hipMemset(d_centroidOct,0,3*Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_cTheta,4*vox.nGrain*sizeof(double)));
  nBlocks=Ntot/nThreads;  
  createBasePlateGrains<<<nBlocks,nThreads>>>(d_vox,d_gID,d_vState,d_g,d_Sites,d_extents, 
					    d_centroidOct,Ntot);
  nThreads=128;
  nBlocks=vox.nGrain/nThreads;
  hipDeviceSynchronize();
  createBasePlateOrientations<<<nBlocks,nThreads>>>(d_vox,d_cTheta,d_g);
  HandleError( hipPeekAtLastError() );
  hipDeviceSynchronize();
  hipFree(d_Sites);
  Sites.clear();
  Sites.shrink_to_fit();
  // end initialize and create baseplate
  //-----------------------------------------------


  //-----------------------------------------------
  // run simulation loop 
  int indout,nlayerTot,npg,nbuf1,nbuf2;
  nlayerTot=int(ceil( (double)(g.nX[2]-g.Nzhg)/(double)g.nZlayer));

  // addlayer and update laser
  vox.AddLayer1Macro(d_vox,g,d_g,&d_cTheta,d_centroidOct,d_gID,d_vState,nbuf2);
  UpdateLaserGlobal<<<1,1>>>(d_g,d_lcoor,d_lcoor2);

  
  HandleError(hipMemcpy(&(vox.nGrain), &(d_vox->nGrain), sizeof(int), hipMemcpyDeviceToHost));
  HandleError(hipMemcpy(vox.gID, d_gID, Ntot*sizeof(int), hipMemcpyDeviceToHost));
  HandleError(hipMemcpy(vox.cTheta, d_cTheta, nbuf2*sizeof(double), hipMemcpyDeviceToHost));
  filout="tmp";
  vox.WriteToHDF1(filout, g, TempF.TempCurr);



  
  //while (!g.bcheck){
  TempF.tInd = int(round(g.time/TempF.DelT));
  calcTemptInd<<<1,1>>>(d_g,d_TempF);
  nThreads=512; nBlocks=Ntot/nThreads;
  TempF.AnalyticalTempCalcMacro(d_g,d_TempF, d_Tempvals,d_lcoor,d_lcoor2,d_ispvec,
                                        nThreads, nBlocks, Ntot);
  // call global temp analytic
  // call global updatevoxels; 
  UpdateLaserGlobal<<<1,1>>>(d_g,d_lcoor,d_lcoor2);
  HandleError(hipMemcpy(&g.bcheck,&(d_g->bcheck),sizeof(int),hipMemcpyDeviceToHost));
  HandleError(hipMemcpy(&g.inewlayerflg,&(d_g->inewlayerflg),sizeof(int),hipMemcpyDeviceToHost));


  if (g.inewlayerflg==1){ 
    HandleError(hipMemcpy(&(vox.nGrain), &(d_vox->nGrain), sizeof(int), hipMemcpyDeviceToHost));
    vox.CleanLayerMacro(g,d_vox,d_gID,&d_cTheta);
  }    
    // cpu: assign indoutput and if statement 
    // if true, then do memcpy of necessary arrays and write out hdf5


    // right now g.updatetime is here, think about putting it in update laser
  g.UpdateTime2(TempF.DelT);
  UpdateTime2Global<<<1,1>>>(d_g,TempF.DelT);
  if (g.inewlayerflg==1){
    vox.AddLayer1Macro(d_vox,g,d_g,&d_cTheta,d_centroidOct,d_gID,d_vState,nbuf2);
  }



  //}





  // end run simulation loop 
  //-----------------------------------------------

  //free cpu memory

  //free memory gpu

  hipFree(d_lcoor);
  hipFree(d_lcoor2);
  //hipFree(d_neighID);
  //hipFree(d_neighptr);
  hipFree(d_Tempvals);
  hipFree(d_ispvec);
  hipFree(d_gID);
  hipFree(d_vState);
  hipFree(d_extents);
  hipFree(d_centroidOct);
  hipFree(d_cTheta);
  hipFree(d_vox);
  hipFree(d_g);
  hipFree(d_TempF);
  
  
  return 0;
}
