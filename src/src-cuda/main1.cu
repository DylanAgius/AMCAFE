#include "hip/hip_runtime.h"

#include "Grid.cuh"
#include "VoxelsCA.cuh"
#include "BasePlate.cuh"
#include "TempField.cuh"
#include "iostream"
#include "vector"
#include <math.h>
#include <chrono>
#include <thread>
#include <algorithm>
#include "fstream"

static void HandleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main(int argc, char *argv[])
{
  //-----------------------------------------------
  // initialize and create base plate
  // set up all pointers for arrays in class
  double *d_lcoor,*d_lcoor2;
  // voxels
  int *d_gID,*d_ineighID,*d_neighptr,*d_vState,nBlocks,nThreads;
  double *d_cTheta,*d_extents,*d_centroidOct;
  // tempfield
  double *d_Temp,*d_ispvec;
  // initialize class variables
  auto texec1 = std::chrono::high_resolution_clock::now();
  std::string filbaseOut,filout,filLogOut,filParamIn;
  filParamIn = argv[1];
  Grid g(filParamIn);
  Grid *d_g;
  HandleError(hipMallocManaged((void**)&d_g,sizeof(Grid)));
  HandleError(hipMemcpy(d_g, &g, sizeof(Grid), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_lcoor,2*g.NpT*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_lcoor2,2*g.NpT*sizeof(double)));
  HandleError(hipMemcpy(d_lcoor,(g.lcoor), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  HandleError(hipMemcpy(d_lcoor2,(g.lcoor2), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  TempField TempF(g);
  TempF.InitializeAnalytic(g);
  TempField *d_TempF;
  HandleError(hipMallocManaged((void**)&d_TempF,sizeof(Grid)));
  HandleError(hipMemcpy(d_TempF, &TempF, sizeof(TempField), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_ispvec,g.NpT*sizeof(int)));
  HandleError(hipMemcpy(d_ispvec,(TempF.ispvec), g.NpT*sizeof(int), hipMemcpyHostToDevice));
  std::vector<double> bpSites;
  GenerateGrainSites(g,bpSites);
  VoxelsCA vox(g);
  vox.nGrain = bpSites.size()/3;
  double *d_bpSites;
  HandleError(hipMallocManaged((void**)&d_bpSites,vox.nGrain*3*sizeof(double)));
  HandleError(hipMemcpy(d_bpSites,bpSites.data(), vox.nGrain*3*sizeof(double), hipMemcpyHostToDevice));
  vox.cTheta=(double*)malloc(vox.nGrain*4*sizeof(double));
  VoxelsCA *d_vox;
  HandleError(hipMallocManaged((void**)&d_vox,sizeof(VoxelsCA)));
  HandleError(hipMemcpy(d_vox, &vox, sizeof(VoxelsCA), hipMemcpyHostToDevice));
  nThreads=1024;
  int Ntot=g.nX[0]*g.nX[1]*g.nX[2];
  HandleError(hipMallocManaged((void**)&d_gID,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_vState,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_extents,Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_centroidOct,3*Ntot*sizeof(double)));
  HandleError(hipMemset(d_gID,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_vState,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_extents,0,Ntot*sizeof(double)));
  HandleError(hipMemset(d_centroidOct,0,3*Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_cTheta,4*vox.nGrain*sizeof(double)));
  nBlocks=Ntot/nThreads;  
  createBasePlateGrains<<<nBlocks,nThreads>>>(d_vox,d_gID,d_vState,d_g,d_bpSites,d_extents, 
					    d_centroidOct,Ntot);
  nThreads=128;
  nBlocks=vox.nGrain/nThreads;
  hipDeviceSynchronize();
  createBasePlateOrientations<<<nBlocks,nThreads>>>(d_vox,d_cTheta);
  HandleError( hipPeekAtLastError() );
  hipDeviceSynchronize();
  hipFree(d_bpSites);
  bpSites.clear();
  bpSites.shrink_to_fit();
  // end initialize and create baseplate
  //-----------------------------------------------

  HandleError(hipMemcpy(vox.gID, d_gID, Ntot*sizeof(int), hipMemcpyDeviceToHost));
  HandleError(hipMemcpy(vox.cTheta, d_cTheta, 4*vox.nGrain*sizeof(double), hipMemcpyDeviceToHost));
  filout="tmp";
  vox.WriteToHDF1(filout, g, TempF.TempCurr);


  //-----------------------------------------------
  // run simulation loop 
  int indout,nlayerTot;
  nlayerTot=int(ceil( (double)(g.nX[2]-g.Nzhg)/(double)g.nZlayer));


  // call addlayer and add flg to call updatelaser within

  while (!g.bcheck){
    // instead of temp.tind use cpu tind (its only used for outputting)
    // call global temp analytic
    // call global updatevoxels; within update voxels call updatelaser
                // (add an update bcheck in updatelaser)

    // do a memcpy of Grid object from device to cpu

    // if (g.inewlayerflg==1){ call global vox.CleanLayer();}
    
    // cpu: assign indoutput and if statement 
    // if true, then do memcpy of necessary arrays and write out hdf5


    // right now g.updatetime is here, think about putting it in update laser

    // if (g.inewlayerflg==1){call global vox.AddLayer1();}

  }





  // end run simulation loop 
  //-----------------------------------------------

  //free cpu memory

  //free memory gpu

  hipFree(d_lcoor);
  hipFree(d_lcoor2);
  hipFree(d_neighID);
  hipFree(d_neighptr);
  hipFree(d_Temp);
  hipFree(d_sipvec);
  hipFree(d_gID);
  hipFree(d_vState);
  hipFree(d_extents);
  hipFree(d_centroidOct);
  hipFree(d_cTheta);
  hipFree(d_vox);
  hipFree(d_g);
  hipFree(d_TempF);
  
  
  return 0;
}
