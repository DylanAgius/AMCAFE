#include "hip/hip_runtime.h"


#include "Grid.cuh"
#include "VoxelsCA.cuh"
#include "BasePlate.cuh"
#include "TempField.cuh"
#include "iostream"
#include "vector"
#include <math.h>
#include <chrono>
#include <thread>
#include <algorithm>
#include "fstream"



static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HandleError( err ) (HandleError( err, __FILE__, __LINE__ ))


int main(int argc, char *argv[])
{
  //-----------------------------------------------
  // initialize and create base plate
  // set up all pointers for arrays in class
  auto texec1 = std::chrono::high_resolution_clock::now();
  double *d_lcoor,*d_lcoor2;
  // voxels
  int *d_gID,*d_vState,nBlocks,
    nThreads,*d_ispvec;
  double *d_cTheta,*d_extents,*d_centroidOct;
  // tempfield
  double *d_Tempvals;
  // initialize class variables
  std::string filbaseOut,filout,filLogOut,filParamIn;
  filParamIn = argv[1];
  filbaseOut = "CA3D"+filParamIn.substr(0,filParamIn.find("."));
  filLogOut="CA3D"+filParamIn.substr(0,filParamIn.find("."))+".log";
  Grid g(filParamIn);
  Grid *d_g;
  int Ntot=g.nX[0]*g.nX[1]*g.nX[2];
  HandleError(hipMallocManaged((void**)&d_g,sizeof(Grid)));
  HandleError(hipMemcpy(d_g, &g, sizeof(Grid), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_lcoor,2*g.NpT*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_lcoor2,2*g.NpT*sizeof(double)));
  HandleError(hipMemcpy(d_lcoor,(g.lcoor), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  HandleError(hipMemcpy(d_lcoor2,(g.lcoor2), 2*g.NpT*sizeof(double), hipMemcpyHostToDevice));
  TempField TempF(g);
  TempF.InitializeAnalytic(g);
  TempField *d_TempF;
  HandleError(hipMallocManaged((void**)&d_TempF,sizeof(Grid)));
  HandleError(hipMalloc((void**)&d_Tempvals,Ntot*sizeof(double)));
  HandleError(hipMemcpy(d_TempF, &TempF, sizeof(TempField), hipMemcpyHostToDevice));
  HandleError(hipMallocManaged((void**)&d_ispvec,g.NpT*sizeof(int)));
  HandleError(hipMemcpy(d_ispvec,(TempF.ispvec), g.NpT*sizeof(int), hipMemcpyHostToDevice));
  std::vector<double> Sites;
  GenerateGrainSites(g,Sites);
  VoxelsCA vox(g);
  vox.nGrain = Sites.size()/3;
  double *d_Sites;
  HandleError(hipMallocManaged((void**)&d_Sites,vox.nGrain*3*sizeof(double)));
  HandleError(hipMemcpy(d_Sites,Sites.data(), vox.nGrain*3*sizeof(double), hipMemcpyHostToDevice));
  vox.cTheta=(double*)malloc(vox.nGrain*4*sizeof(double));
  VoxelsCA *d_vox;
  HandleError(hipMallocManaged((void**)&d_vox,sizeof(VoxelsCA)));
  HandleError(hipMemcpy(d_vox, &vox, sizeof(VoxelsCA), hipMemcpyHostToDevice));
  nThreads=1024;
  HandleError(hipMallocManaged((void**)&d_gID,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_vState,Ntot*sizeof(int)));
  HandleError(hipMallocManaged((void**)&d_extents,Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_centroidOct,3*Ntot*sizeof(double)));
  HandleError(hipMemset(d_gID,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_vState,0,Ntot*sizeof(int)));
  HandleError(hipMemset(d_extents,0,Ntot*sizeof(double)));
  HandleError(hipMemset(d_centroidOct,0,3*Ntot*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_cTheta,4*vox.nGrain*sizeof(double)));
  nBlocks=Ntot/nThreads;  
  createBasePlateGrains<<<nBlocks,nThreads>>>(d_vox,d_gID,d_vState,d_g,d_Sites,d_extents, 
					    d_centroidOct,Ntot);
  nThreads=128;
  nBlocks=vox.nGrain/nThreads;
  hipDeviceSynchronize();
  createBasePlateOrientations<<<nBlocks,nThreads>>>(d_vox,d_cTheta,d_g);
  HandleError( hipPeekAtLastError() );
  hipDeviceSynchronize();
  hipFree(d_Sites);
  Sites.clear();
  Sites.shrink_to_fit();
  // end initialize and create baseplate
  //-----------------------------------------------


  //-----------------------------------------------
  // run simulation loop 
  int indOut,nbuf;
  std::ofstream fplog;
  fplog.open(filLogOut.c_str());
  fplog << "Time index= ,Total clock time passed(s)"<<std::endl;
  // addlayer and update laser
  hipDeviceSynchronize();
  vox.AddLayer1Macro(d_vox,g,d_g,&d_cTheta,d_centroidOct,d_gID,d_vState,nbuf);
  HandleError( hipPeekAtLastError() );
  UpdateLaserGlobal<<<1,1>>>(d_g,d_lcoor,d_lcoor2);
  HandleError( hipPeekAtLastError() );
  int cc=0;
  while (!g.bcheck){
  TempF.tInd = int(round(g.time/TempF.DelT));
  calcTemptInd<<<1,1>>>(d_g,d_TempF);
  HandleError( hipPeekAtLastError() );
  nThreads=512; nBlocks=Ntot/nThreads;
  // call global temp analytic
  hipDeviceSynchronize();
  TempF.AnalyticalTempCalcMacro(d_g,d_TempF, d_Tempvals,d_lcoor,d_lcoor2,d_ispvec,
                                        nThreads, nBlocks, Ntot);
  HandleError( hipPeekAtLastError() );
  // call global updatevoxels;
  UpdateLaserGlobal<<<1,1>>>(d_g,d_lcoor,d_lcoor2);
  HandleError( hipPeekAtLastError() );
  HandleError(hipMemcpy(&g.bcheck,&(d_g->bcheck),sizeof(bool),hipMemcpyDeviceToHost));
  HandleError(hipMemcpy(&g.inewlayerflg,&(d_g->inewlayerflg),sizeof(int),hipMemcpyDeviceToHost));
  if (g.inewlayerflg==1){ 
    HandleError(hipMemcpy(&(vox.nGrain), &(d_vox->nGrain), sizeof(int), hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    vox.CleanLayerMacro(d_vox,d_gID,&d_cTheta,Ntot);
    HandleError( hipPeekAtLastError() );
  }    
  
  indOut = TempF.tInd % g.outint;

  if (indOut==0 || g.bcheck || (g.inewlayerflg==1 && g.outNL==0)){
    // if true, then do memcpy of necessary arrays and write out hdf5
    // (note nGrain is copied over from device in CleanLayerMacro
    nbuf=4*vox.nGrain;
    resizeArray(&(vox.cTheta),nbuf);
    HandleError(hipMemcpy(vox.gID, d_gID, Ntot*sizeof(int), hipMemcpyDeviceToHost));
    HandleError(hipMemcpy(vox.vState, d_vState, Ntot*sizeof(int), hipMemcpyDeviceToHost));
    HandleError(hipMemcpy(TempF.TempCurr, d_Tempvals, Ntot*sizeof(double), hipMemcpyDeviceToHost));
    HandleError(hipMemcpy(vox.cTheta, d_cTheta, 4*vox.nGrain*sizeof(double), hipMemcpyDeviceToHost));
    filout = filbaseOut+std::to_string(TempF.tInd);
    hipDeviceSynchronize();
    vox.WriteToHDF1(filout, g, TempF.TempCurr);
  } // if (indOut==0 ...
    g.UpdateTime2(TempF.DelT);
  UpdateTime2Global<<<1,1>>>(d_g,TempF.DelT);
  HandleError( hipPeekAtLastError() ); 
  if (g.inewlayerflg==1){
    hipDeviceSynchronize();
    vox.AddLayer1Macro(d_vox,g,d_g,&d_cTheta,d_centroidOct,d_gID,d_vState,nbuf);
    HandleError( hipPeekAtLastError() );
  }
  auto texec2 = std::chrono::high_resolution_clock::now();
  auto delTexec = std::chrono::duration_cast<std::chrono::seconds>( texec2 - texec1 ).count();
  fplog << TempF.tInd<<","<<delTexec<<std::endl;

  }
  hipDeviceSynchronize();
  fplog.close();



  // end run simulation loop 
  //-----------------------------------------------

  //free cpu memory

  //free memory gpu

  hipFree(d_lcoor);
  hipFree(d_lcoor2);
  //hipFree(d_neighID);
  //hipFree(d_neighptr);
  hipFree(d_Tempvals);
  hipFree(d_ispvec);
  hipFree(d_gID);
  hipFree(d_vState);
  hipFree(d_extents);
  hipFree(d_centroidOct);
  hipFree(d_cTheta);
  hipFree(d_vox);
  hipFree(d_g);
  hipFree(d_TempF);
  
  
  return 0;
}
