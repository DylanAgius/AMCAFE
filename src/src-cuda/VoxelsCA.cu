#include "hip/hip_runtime.h"
// define member functions of VoxelCA

#include "Grid.cuh"
#include "VoxelsCA.cuh"
#include "iostream"
#include "fstream"
#include "sstream"
#include <math.h>
#include <algorithm>
#include <random>
#include <ctime>
#include "SampleOrientation.cuh"
#include <chrono>
#include <adios2.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

static void HandleError(hipError_t err) {
    if (err != hipSuccess) {
        printf("%s \n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void getSites(Grid *g,VoxelsCA *vx,double *xs, int numPG)
{
// calculates sites for Voronoi (done in own global function to avoid 
// inter-block race condition)
  int tid=threadIdx.x+blockDim.x*blockIdx.x, subsq=0,iz1,js,
    stride=blockDim.x*gridDim.x;
  unsigned int seedL = (vx->seed0+tid*100+g->tInd*10) % 4294967295;
  hiprandState_t s1;
  hiprand_init(seedL,subsq,0,&s1);
  double Lx=g->nX[0]*g->dX[0], Ly=g->nX[1]*g->dX[1],
    lt=g->layerT,zloc0;
  iz1 = g->ilaserLoc - g->nZlayer;
  zloc0 = iz1*g->dX[0];
  js=tid;
  while (js <numPG){
    xs[3*js]= hiprand_uniform(&s1)*Lx;
    xs[3*js+1]= hiprand_uniform(&s1)*Ly;
    xs[3*js+2]= hiprand_uniform(&s1)*lt+zloc0;
    js+=stride;
  }
}
__global__ void addLayer1Part1(Grid *g,VoxelsCA *vx,double *xs, double *troids,
			  int *gD, int *vs, int *itmp, int numPG)
{
  int tid=threadIdx.x+blockDim.x*blockIdx.x,iz1,js,
    stride=blockDim.x*gridDim.x,j1,j2,j3,nX1=g->nX[0],
    iz2=g->ilaserLoc,i3=g->nX[0]*g->nX[1],i2,ng1=vx->nGrain;
  double dx=g->dX[0],dsqc,dsq,xc,yc,zc;
  iz1 = g->ilaserLoc - g->nZlayer;
  js=tid+i3*iz1;
  while (js < i3*iz2){
    j3 = js/i3; // note that int/int is equivalent to floor                                                                         
    j2 = (js - i3*j3)/(nX1);
    j1 = js - i3*j3 - nX1*j2;
    zc = (j3+.5)*dx;;
    yc = (j2+.5)*dx;
    xc = (j1+.5)*dx;
    dsqc=1e6;
    for (int jz=0;jz<numPG;++jz){
      dsq = pow(xc - xs[3*jz],2.)+pow(yc-xs[3*jz+1],2.)+pow(zc-xs[3*jz+2],2.);
      if (dsq<dsqc){i2=jz; dsqc=dsq;}
    } // for (int jz
    gD[js] = i2+1+ng1;
    itmp[i2]=1;
    vs[js] = 3;
    troids[3*js] = xc;
    troids[3*js+1] = yc;
    troids[3*js+2] = zc;
    js+=stride;
  } // while (js < i3*...
} // end addlayer1part1

__global__ void addLayer1Part2(Grid *g,VoxelsCA *vx, double *dctheta,
			  int *gD, int *itmp, int numPG)
{

  // !!***************************************************
  // FUNCTION CAN ONLY RUN WITH 1 BLOCK. OTHERWISE, BUG
  // !!***************************************************

  int tid=threadIdx.x+blockDim.x*blockIdx.x, subsq=0,iz1,js,
    stride=blockDim.x*gridDim.x,ng1=vx->nGrain;
  unsigned int seedL = (vx->seed0+tid*100+g->tInd*10) % 4294967295;
  hiprandState_t s1;
  hiprand_init(seedL,subsq,0,&s1);
  double lt=g->layerT;
  iz1 = g->ilaserLoc - g->nZlayer;
  int i3=g->nX[0]*g->nX[1],i2,i1;
  // ensures a continuous numbering of grain ids
  __shared__ int i2s;
  if (tid==0){
    i2=0;
    for (int j=0;j<numPG;++j){
      if (itmp[j]==1){
	itmp[j]=i2;
	i2+=1;
      }
    }
    i2s=i2;
    vx->nGrain += i2+1;
  } // if (tid==0
  __syncthreads();
  if (i2s != numPG){
    js=tid+i3*iz1;
    while (js < i3*(lt+iz1)){
      i1=gD[js]-1;
      gD[js]=itmp[i1]+1+ng1;
      js+=stride;
    }
  }
  // randomly generate crystallographic orientations 
  double axAng[4];
  i2s+=1;
  js=tid;
  while (js<i2s){
    GenerateSamples(1,seedL,subsq,s1, axAng);
    dctheta[4*(js+ng1)]=axAng[0];
    dctheta[4*(js+ng1)+1]=axAng[1];
    dctheta[4*(js+ng1)+2]=axAng[2];
    dctheta[4*(js+ng1)+3]=axAng[3];
    js+=stride;
  }
} // __global__ void addlayer1part2

__global__ void addLayer1Part3(const Grid *g,int *gD, int *vs) 
{
  int tid=threadIdx.x+blockDim.x*blockIdx.x,js,i1,ntot,stride;
  stride=blockDim.x*gridDim.x;
  i1=g->nX[0]*g->nX[1]*g->ilaserLoc;
  js=tid+i1;
  ntot=g->nX[0]*g->nX[1]*g->nX[2];
  while (js<ntot){
    vs[js]=0;
    gD[js]=0;
    js+=stride;
  }
} // __global__ void addlayer1par3


__global__ void copyGlobal(double *x1,double *x0, int n)
{
  int tid=threadIdx.x + blockDim.x*blockIdx.x,js,stride;
  js=tid;
  stride=blockDim.x*gridDim.x;
  while (js < n){
    x1[js] = x0[js];
    js +=stride;
  }
}

__global__ void cleanLayerPart1(VoxelsCA *dvx,int *dgid,int *gvolflg, int *itmp,int Ntot)
{
  int tid=threadIdx.x+blockIdx.x*blockDim.x,js,i1,stride;
  stride = blockDim.x*gridDim.x;
  js=tid;
  while (js < Ntot){
    i1=dgid[js]-1;
    gvolflg[i1]=1;
    js+=stride;
  }
}
__global__ void cleanLayerPart2(VoxelsCA *dvx,int *gvolflg, int *itmp)
{
  int ng=dvx->nGrain,tid=threadIdx.x+blockIdx.x*blockDim.x,i2;
  if (tid==0){
    i2=0;
    for (int j=0;j<ng;++j){
      if (gvolflg[j]>0){
	itmp[j] = i2+1;
	i2+=1;
      }
    }
    gvolflg[ng]=i2;
  } 
}

__global__ void cleanLayerPart3(VoxelsCA *dvx,int *dgid,int *gvolflg, int *itmp,
				double *ctmp, double *dcth,int Ntot)
{
  int ng,tid=threadIdx.x+blockIdx.x*blockDim.x,js,stride,i1;
  stride=blockDim.x*gridDim.x;
  js=tid;
  ng=dvx->nGrain;
  while(js<ng){
    if (gvolflg[js]>0){
      ctmp[4*(itmp[js]-1)] = dcth[4*js];
      ctmp[4*(itmp[js]-1)+1] = dcth[4*js+1];
      ctmp[4*(itmp[js]-1)+2] = dcth[4*js+2];
      ctmp[4*(itmp[js]-1)+3] = dcth[4*js+3];
    }
    js+=stride;
  }
  js=tid;
  while (js<Ntot){
    if (dgid[js]!=0){
      i1 = itmp[dgid[js]-1];
      dgid[js]=i1;
    }
    js+=stride;
  }
}
__global__ void cleanLayerPart4(VoxelsCA *dvx, int *gvolflg)
{
  int tid=threadIdx.x+blockDim.x*blockIdx.x;
  int ng = dvx->nGrain;
  if (tid==0){dvx->nGrain = gvolflg[ng];}
}
__global__ void convertSolid1Part1(Grid *dg,int *vst,double *exts,int ntot)
{
  // ONLY RUN WITH 1 BLOCK
  int inei[27],vnei[26],js,nb,
    tid=threadIdx.x,jz,stride=blockDim.x;
  bool issol;
  extern __shared__ bool i1[];
  nb=ntot/stride + 1;
  for (int j=0;j<nb;++j){
    js = tid+j*stride;
    __syncthreads();
    i1[tid]=0;
    if (js<ntot){
      if (vst[js]==2){
	dg->GetNeighbors(js,inei);
	for (int j1=0;j1<inei[26];++j1){vnei[j1]=vst[inei[j1]];}
	jz=0;
	issol=1;
	while (issol && (jz<inei[26])){
	  issol = issol && (vnei[jz]>=2);
	  jz+=1;
	}
	if (issol){
	  i1[threadIdx.x]=1;
	  exts[js]=0.0;
	}
      }
    }
    __syncthreads();
    if (i1[tid]){vst[js]=3;}
  }
}
__global__ void convertSolid1Part2(Grid *dg,int *vst,int ntot)
{
  // ONLY RUN WITH 1 BLOCK 
  int inei[27],vnei[26],js,nb,
    tid=threadIdx.x,stride=blockDim.x;
  extern __shared__ bool i1[];
  nb=ntot/stride+1;
  for (int j=0;j<nb;++j){
    js=tid+j*stride;
    __syncthreads();
    i1[threadIdx.x]=0;
    inei[26]=0;
    if (js<ntot){
      if (vst[js]==1){
	dg->GetNeighbors(js,inei);
	for (int j1=0;j1<inei[26];++j1){vnei[j1]=vst[inei[j1]];}
      }
    }
    __syncthreads();
    for (int j1=0;j1<inei[26];++j1){
      if (vnei[j1]==3){vst[inei[j1]]=2;}
    }
  }
}
__global__ void setLiquid3(Grid *dg,int *dgid,int *dvstate,double *dtempval, double *dexts)
{
  // makes cell liquid if temperature exceeds liquidus
  int n1=dg->nX[0]*dg->nX[1],ntot,iz1,js,
    tid=threadIdx.x+blockIdx.x*blockDim.x,stride=blockDim.x*gridDim.x,
    tL=dg->tL;
  ntot=n1*dg->nX[2];
  iz1=dg->ilaserLoc*n1;
  js=tid;
  while (js<ntot){
    if (dtempval[js]>=tL){
      if (js < n1){
	dvstate[js]=2;
      } else if (js < iz1){
	dvstate[js]=1;
	dgid[js]=0; 
	dexts[js] = 0.0;
      }
    }
    js+=stride;
  }
}
__global__ void updateVoxelsPart1(int *vstate,int *vs2cc,int ntot)
{
  int tid=threadIdx.x+blockIdx.x*blockDim.x,js,stride=blockDim.x*gridDim.x,s,
    tidL=threadIdx.x,bidL=blockIdx.x,nthread=blockDim.x;
  extern __shared__ volatile int sh[];
  js=tid;
  sh[tidL]=0;
  while (js<ntot){
    if (vstate[js]==2){sh[tidL]+=1;}
    js+=stride;
  }
  __syncthreads();
  s=nthread;
  while (s>=128){
    if (nthread >= s){
      if (tidL<s/2){sh[tidL] += sh[tidL+s/2];}
      __syncthreads();
    }
    s/=2;
  }
  if (tidL<32) {
    if (nthread >=64) {sh[tidL] += sh[tidL+32];}
    if (nthread >=32) {sh[tidL] += sh[tidL+16];}
    if (nthread >=16) {sh[tidL] += sh[tidL+8];}
    if (nthread >= 8) {sh[tidL] += sh[tidL+4];}
    if (nthread >= 4) {sh[tidL] += sh[tidL+2];}
    if (nthread >= 2) {sh[tidL] += sh[tidL+1];}
  }
  if (tidL==0) {
    vs2cc[bidL] = sh[tidL];
  }
}

__global__ void updateVoxelsPart2(int *vs2cc, int *vs2, bool *disf,double *dtinc)
{
  vs2[0]=vs2cc[0];
  vs2[1]=0;
  dtinc[0]=0.0;
  disf[0]=1; // initialize to true (false exits voxel growth loop)
}

__global__ void updateVoxelsPart3(Grid *dg,int *dgid,int *dvstate, double *dctheta,double *dtempval,
				  double *dctroid,double *dexts,double *dtinc,int *vs2,bool *disf,
				  float *dtminG,int *jindG,int *j1indG,int ntot)
{
  int tid=threadIdx.x,gid=threadIdx.x+blockIdx.x*blockDim.x,inei[27],js,
    stride=gridDim.x*blockDim.x,jx[3],timeUntil,dr,i3=dg->nX[0]*dg->nX[1],
    i2=dg->nX[0],s,nthread=blockDim.x;
  double vhat,dnx[3],dlocX[3],omega,ax[3],rRot[3][3],th,ph,tmelt=dg->tL,
    Avel=dg->Avel,nvel=dg->nvel,dx=dg->dX[0];
  bool isaxt,isaxv;
  extern __shared__ volatile int sh[];
  volatile int *jvec=sh;
  volatile int *j1vec=&jvec[nthread];
  volatile float *dtminv=(float*)&j1vec[nthread];
  disf[0]=(vs2[0]!=vs2[1] && dtinc[0]<dg->bmDelT);
  if (disf[0]){
    dtminv[tid]=1e6;
    if (tid==0){vs2[1]=vs2[0];}
    js=gid;
    while (js<ntot){
      if (dvstate[js]==2){
	dg->GetNeighbors(js,inei);
	isaxt=1;
	isaxv=1;
	for (int j=0;j<inei[26];++j){
	  isaxt=isaxt && dtempval[inei[j]]<tmelt;
	  isaxv=isaxv && dvstate[inei[j]]!=1;
	}
      	if ( (!isaxv) && isaxt){
	  omega=dctheta[4*(dgid[js]-1)];
	  ax[0]=dctheta[4*(dgid[js]-1)+1];
	  ax[1]=dctheta[4*(dgid[js]-1)+2];
	  ax[2]=dctheta[4*(dgid[js]-1)+3];
	  loadRotMat(omega,ax,rRot);
	  dtempval[js]>=tmelt ? vhat=0. : vhat = Avel*pow(tmelt-dtempval[js],nvel);
          for (int j=0;j<inei[26];++j){
            if (dvstate[inei[j]] != 1 ){continue;}
            jx[2] = inei[j]/i3;
            jx[1] = (inei[j]- i3*jx[2])/i2;
            jx[0] = inei[j] - i3*jx[2] - i2*jx[1];      
            dnx[0] = (double(jx[0])+.5)*dx - dctroid[3*js];
            dnx[1] = (double(jx[1])+.5)*dx - dctroid[3*js+1];
            dnx[2] = (double(jx[2])+.5)*dx - dctroid[3*js+2];
            th = atan2(fabs(dnx[1]),fabs(dnx[0]));
            th > HIP_PI/4.0 ? th= HIP_PI/2.0 - th: th ;
            ph = atan2(pow(pow(dnx[0],2.0)+pow(dnx[1],2.0),.5),fabs(dnx[2]));
            ph < HIP_PI/4.0 ? ph = HIP_PI/2.0 - ph: ph ;
            // matrix is local->global; need to multiply by transpose for global->local
            // put into 1st quadrant b/c of symmetry
            dlocX[0] = fabs(rRot[0][0]*dnx[0]+rRot[1][0]*dnx[1]+rRot[2][0]*dnx[2]);
            dlocX[1] = fabs(rRot[0][1]*dnx[0]+rRot[1][1]*dnx[1]+rRot[2][1]*dnx[2]);
            dlocX[2] = fabs(rRot[0][2]*dnx[0]+rRot[1][2]*dnx[1]+rRot[2][2]*dnx[2]);
            dr = pow(cos(th)*sin(ph),.5)*(dlocX[0]+dlocX[1]+dlocX[2]) - dexts[js];
            timeUntil = dr/vhat;
	    if (timeUntil<dtminv[tid]){
	      dtminv[tid] = timeUntil;
	      jvec[tid] = js;
	      j1vec[tid] = inei[j];
	    }
	  } // for (int j=0
	} // if ((!isaxv) && isaxt...
      }	// if (dvstate[js]==2...
      js+=stride;
    } // while (js<ntot
    // reduce within block    
    __syncthreads();
    s=nthread;
    while (s>=128){
      if (nthread >= s){
	if (tid<s/2){
	  if (dtminv[tid]>dtminv[tid+s/2]){
	    dtminv[tid] = dtminv[tid+s/2];
	    jvec[tid] = jvec[tid+s/2];
	    j1vec[tid] = j1vec[tid+s/2];
	  }
	}
	__syncthreads();
      }
      s/=2;
    }
    if (tid<32) {
      if (nthread >=64) {
	if (dtminv[tid]> dtminv[tid+32]){
	  dtminv[tid] = dtminv[tid+32];
	  jvec[tid] = jvec[tid+32];
	  j1vec[tid] = j1vec[tid+32];
	}
      }
      if (nthread >=32) {
	if (dtminv[tid]> dtminv[tid+16]){
	  dtminv[tid] = dtminv[tid+16];
	  jvec[tid] = jvec[tid+16];
	  j1vec[tid] = j1vec[tid+16];
	}
      }
      if (nthread >=16) {
	if (dtminv[tid]> dtminv[tid+8]){
	  dtminv[tid] = dtminv[tid+8];
	  jvec[tid] = jvec[tid+8];
	  j1vec[tid] = j1vec[tid+8];
	}
      }
      if (nthread >= 8) {
	if (dtminv[tid]> dtminv[tid+4]){
	  dtminv[tid] = dtminv[tid+4];
	  jvec[tid] = jvec[tid+4];
	  j1vec[tid] = j1vec[tid+4];
	}
      }
      if (nthread >= 4) {
	if (dtminv[tid]> dtminv[tid+2]){
	  dtminv[tid] = dtminv[tid+2];
	  jvec[tid] = jvec[tid+2];
	  j1vec[tid] = j1vec[tid+2];
	}
      }
      if (nthread >= 2) {
	if (dtminv[tid]> dtminv[tid+1]){
	  dtminv[tid] = dtminv[tid+1];
	  jvec[tid] = jvec[tid+1];
	  j1vec[tid] = j1vec[tid+1];
	}
      }
    }  // if (tid<32) ...
    if (tid==0) {
      dtminG[blockIdx.x] = dtminv[tid];
      jindG[blockIdx.x] = jvec[tid];
      j1indG[blockIdx.x] = j1vec[tid];
    }
  } // if (disf[0])
}

__global__ void reduceVoxelCapture(float *dtminG,int *jindG, int *j1indG, int n)
{
  int js,tid=threadIdx.x+blockDim.x*blockIdx.x,stride=blockDim.x*gridDim.x,s,
    nthread=blockDim.x,tidL=threadIdx.x,bidL=blockIdx.x;
  extern __shared__ volatile int sh[];
  volatile int *jvec=sh;
  volatile int *j1vec=&jvec[nthread];
  volatile float *dtminv=(float*)&j1vec[nthread];
  js=tid;
  dtminv[tidL]=1e6;
  while (js<n){
    if (dtminv[tidL]>dtminG[js]){
      dtminv[tidL] = dtminG[js];
      jvec[tidL] = jindG[js];
      j1vec[tidL] = j1indG[js];
    }      
    js+=stride;
  }
  __syncthreads();
  s=nthread;
  while (s>=128){
    if (nthread >= s){
      if (tidL<s/2){
	if (dtminv[tidL]>dtminv[tidL+s/2]){
	  dtminv[tidL] = dtminv[tidL+s/2];
	  jvec[tidL] = jvec[tidL+s/2];
	  j1vec[tidL] = j1vec[tidL+s/2];
	}
      }
      __syncthreads();
    }
    s/=2;
  }
  if (tidL<32) {
    if (nthread >=64) {
      if (dtminv[tidL]>dtminv[tidL+32]){
	dtminv[tidL] = dtminv[tidL+32];
	jvec[tidL] = jvec[tidL+32];
	j1vec[tidL] = j1vec[tidL+32];
      }
    }
    if (nthread >=32) {
      if (dtminv[tidL]>dtminv[tidL+16]){
	dtminv[tidL] = dtminv[tidL+16];
	jvec[tidL] = jvec[tidL+16];
	j1vec[tidL] = j1vec[tidL+16];
      }
    }
    if (nthread >=16) {
      if (dtminv[tidL]>dtminv[tidL+8]){
	dtminv[tidL] = dtminv[tidL+8];
	jvec[tidL] = jvec[tidL+8];
	j1vec[tidL] = j1vec[tidL+8];
      }
    }
    if (nthread >= 8) {
      if (dtminv[tidL]>dtminv[tidL+4]){
	dtminv[tidL] = dtminv[tidL+4];
	jvec[tidL] = jvec[tidL+4];
	j1vec[tidL] = j1vec[tidL+4];
      }
    }
    if (nthread >= 4) {
      if (dtminv[tidL]>dtminv[tidL+2]){
	dtminv[tidL] = dtminv[tidL+2];
	jvec[tidL] = jvec[tidL+2];
	j1vec[tidL] = j1vec[tidL+2];
      }
    }
    if (nthread >= 2) {
      if (dtminv[tidL]>dtminv[tidL+1]){
	dtminv[tidL] = dtminv[tidL+1];
	jvec[tidL] = jvec[tidL+1];
	j1vec[tidL] = j1vec[tidL+1];
      }
    }
  } // if (tid<32 ...
  if (tidL==0) {
    dtminG[bidL] = dtminv[tidL];
    jindG[bidL] = jvec[tidL];
    j1indG[bidL] = j1vec[tidL];
  }
}

__global__ void updateVoxelsPart4(Grid *dg,VoxelsCA *dvox,int *dgid,int *dvstate, double *dctheta,
				  double *dctroid,double *dexts,int *vs2,
				  float *dtminG,int *j1indG)
{
  // ONLY RUN WITH 1 THREAD: SERIAL PROCESS
  // this is for nucleation
  int subsq=0,js1,nsamp=1,ng=dvox->nGrain,jx[3],
    i3=dg->nX[0]*dg->nX[1],i2=dg->nX[0];
  unsigned int seedL= (dg->tInd*30+dvox->seed0+64*ng) % 4294967295;
  hiprandState_t s1;
  double dtmin,axAng[4],dx=dg->dX[0];
  dtmin=dtminG[0]; 
  __syncthreads();
  if (dtmin<1e6){
    hiprand_init(seedL,subsq,0,&s1);
    js1=j1indG[0];
    GenerateSamples(nsamp,seedL,subsq,s1,axAng);
    dvstate[js1] = 2;
    vs2[1]+=1;
    dvox->nGrain+=1;
    dctheta[4*ng]=axAng[0];
    dctheta[4*ng+1]=axAng[1];
    dctheta[4*ng+2]=axAng[2];
    dctheta[4*ng+3]=axAng[3];
    dgid[js1]=ng+1;
    jx[2]=js1/i3;
    jx[1]=(js1-i3*jx[2])/i2;
    jx[0]=js1-i3*jx[2]-jx[1];
    dctroid[3*js1]=(double(jx[0])+.5)*dx;
    dctroid[3*js1+1]=(double(jx[1])+.5)*dx;
    dctroid[3*js1+2]=(double(jx[2])+.5)*dx;
    dexts[js1]=0.0;
  }
}
__global__ void updateVoxelsPart5(Grid *dg,int *dvstate, double *dtempval,
				  double *dexts,float *dtminG,int ntot)
{
  // UPDATES EXTENTS 
  int tid=threadIdx.x+blockIdx.x*blockDim.x,js,
    stride=blockDim.x*gridDim.x,inei[27];
  double dtmin,Avel=dg->Avel,nvel=dg->nvel,vhat,tmelt=dg->tL;
  bool isaxt,isaxv;
  dtmin=dtminG[0];
  __syncthreads();
  if (dtmin<1e6){
    js=tid;
    while (js<ntot){
      vhat=0.;
      if (dvstate[js]==2){
	dg->GetNeighbors(js,inei);
	isaxt=1;
	isaxv=1;
	for (int j=0;j<inei[26];++j){
	  isaxt=isaxt && dtempval[inei[j]]<tmelt;
	  isaxv=isaxv && dvstate[inei[j]]!=1;
	}
      	if ( (!isaxv) && isaxt){
	  dtempval[js]>=tmelt ? vhat=0. : vhat = Avel*pow(tmelt-dtempval[js],nvel);
	}	
      }
      dexts[js]+=vhat*dtmin;
      js+=stride;
    }
  }
}

__global__ void updateVoxelsPart6(Grid *dg,VoxelsCA *dvox,int *dgid,int *dvstate, double *dctheta,
				  double *dctroid,double *dexts,int *vs2,double *dtinc,
				  float *dtminG,int *jindG,int *j1indG)
{
  // capture voxel **ONLY USE 1 THREAD - this is serial operation**
  int js,js1,jInd,i3=dg->nX[0]*dg->nX[1],
    i2=dg->nX[0],jx[3],sInd[8][3],jy[3];
  double xI[3],xJ[3],d1I,dI2,d1J,dJ3,L12,L13,l,Lmud,dnorm[3],xiL,
    dx=dg->dX[0],rRot[3][3],dr,th,ph,sdiag0[6][3],sdiag[6][3],dtmin,
    dnx[3],omega,ax[3],locX[3];
      
  dtmin=dtminG[0];
  if (dtmin<1e6){
    js=jindG[0];
    js1=j1indG[0];
    dvstate[js1]=2;
    vs2[1]+=1;
    dgid[js1]=dgid[js];
    jx[2] = js1/i3;
    jx[1] = (js1- i3*jx[2])/i2;
    jx[0] = js1 -i3*jx[2] - i2*jx[1];
    jy[2] = js/i3;
    jy[1] = (js- i3*jy[2])/i2;
    jy[0] = js - i3*jy[2] - i2*jy[1];
    l = pow( pow((jx[0]-jy[0])*dx,2)+ pow((jx[1]-jy[1])*dx,2)+
	     pow((jx[2]-jy[2])*dx,2),.5);
      dnx[0] = (double(jx[0])+.5)*dx - dctroid[3*js];
      dnx[1] = (double(jx[1])+.5)*dx - dctroid[3*js+1];
      dnx[2] = (double(jx[2])+.5)*dx - dctroid[3*js+2];
      omega = dctheta[4*(dgid[js]-1)];
      ax[0] = dctheta[4*(dgid[js]-1)+1];
      ax[1] = dctheta[4*(dgid[js]-1)+2];
      ax[2] = dctheta[4*(dgid[js]-1)+3];
      loadRotMat(omega,ax,rRot);    
      // matrix is local->global; need to multiply by transpose for global->local
      locX[0] = rRot[0][0]*dnx[0]+rRot[1][0]*dnx[1]+rRot[2][0]*dnx[2];
      locX[1] = rRot[0][1]*dnx[0]+rRot[1][1]*dnx[1]+rRot[2][1]*dnx[2];
      locX[2] = rRot[0][2]*dnx[0]+rRot[1][2]*dnx[1]+rRot[2][2]*dnx[2];
      th = atan2(fabs(dnx[1]),fabs(dnx[0]));
      th > HIP_PI/4.0 ? th= HIP_PI/2.0 - th: th;
      ph = atan2(pow(pow(dnx[0],2.0)+pow(dnx[1],2.0),.5),fabs(dnx[2]));
      ph < HIP_PI/4.0 ? ph = HIP_PI/2.0 - ph: ph;
      // signbit returns 0 if positive and 1 if negative
      dr = fabs(locX[0])+fabs(locX[1])+fabs(locX[2]);
      loadS(sdiag0,sInd); 
      for (int j1=0;j1<6;++j1){
	sdiag[j1][0]=sdiag0[j1][0]*dr;
	sdiag[j1][1]=sdiag0[j1][1]*dr;
	sdiag[j1][2]=sdiag0[j1][2]*dr;
      }
      if (locX[2]<0){jx[2]=1;} else{jx[2]=0;}
      if (locX[1]<0){jx[1]=1;} else{jx[1]=0;}
      if (locX[0]<0){jx[0]=1;} else{jx[0]=0;}
      jInd = 4*jx[2]+ 2*jx[1]+ jx[0];
      for (int j1=0;j1<3;++j1){
        dnorm[j1] = pow(locX[0]-sdiag[sInd[jInd][j1]][0],2.0)+
          pow(locX[1]-sdiag[sInd[jInd][j1]][1],2.0)+pow(locX[2]-sdiag[sInd[jInd][j1]][2],2.0);
      } // for (int j1...                                                                                                                                     
      jy[0]=0; jy[2]=0;
      if (dnorm[0]<=dnorm[1] && dnorm[0]<=dnorm[2]){jy[0]=0;}
      if (dnorm[1]<=dnorm[0] && dnorm[1]<=dnorm[2]){jy[0]=1;}
      if (dnorm[2]<=dnorm[0] && dnorm[2]<=dnorm[1]){jy[0]=2;}
      if (dnorm[0]>=dnorm[1] && dnorm[0]>=dnorm[2]){jy[2]=0;}
      if (dnorm[1]>=dnorm[0] && dnorm[1]>=dnorm[2]){jy[2]=1;}
      if (dnorm[2]>=dnorm[0] && dnorm[2]>=dnorm[1]){jy[2]=2;}
      if (jy[0]==jy[2]){
          jy[0]=0; jy[1]=1; jy[2]=2;
      } else {
        jy[1] = 3 - jy[0] - jy[2];
      }
      projectPointLine(locX,&(sdiag[sInd[jInd][jy[0]]][0]),&(sdiag[sInd[jInd][jy[1]]][0]),xI);
      projectPointLine(locX,&(sdiag[sInd[jInd][jy[0]]][0]),&(sdiag[sInd[jInd][jy[2]]][0]),xJ);
      d1I = pow(pow(sdiag[sInd[jInd][jy[0]]][0]-xI[0],2.0) +
                pow(sdiag[sInd[jInd][jy[0]]][1]-xI[1],2.0) +
                pow(sdiag[sInd[jInd][jy[0]]][2]-xI[2],2.0),.5);
      dI2 = pow(pow(sdiag[sInd[jInd][jy[1]]][0]-xI[0],2.0) +
                pow(sdiag[sInd[jInd][jy[1]]][1]-xI[1],2.0) +
                pow(sdiag[sInd[jInd][jy[1]]][2]-xI[2],2.0),.5);
      d1J = pow(pow(sdiag[sInd[jInd][jy[0]]][0]-xJ[0],2.0) +
                pow(sdiag[sInd[jInd][jy[0]]][1]-xJ[1],2.0) +
                pow(sdiag[sInd[jInd][jy[0]]][2]-xJ[2],2.0),.5);
      dJ3 = pow(pow(sdiag[sInd[jInd][jy[2]]][0]-xJ[0],2.0) +
                pow(sdiag[sInd[jInd][jy[2]]][1]-xJ[1],2.0) +
                pow(sdiag[sInd[jInd][jy[2]]][2]-xJ[2],2.0),.5);
      L12 = .5*(fmin(d1I,pow(3.0,.5)*l) + fmin(dI2,pow(3.0,.5)*l) );
      L13 = .5*(fmin(d1J,pow(3.0,.5)*l) + fmin(dJ3,pow(3.0,.5)*l) );
      Lmud =  pow(3.0,.5)* (pow(2.0/3.0,.5)*fmax(L12,L13));
      xiL = 1.0;
      dexts[js1] = pow(cos(th)*sin(ph),.5)*Lmud*xiL;
      dnx[0] = sdiag[sInd[jInd][jy[0]]][0] - Lmud*sdiag0[sInd[jInd][jy[0]]][0];
      dnx[1] = sdiag[sInd[jInd][jy[0]]][1] - Lmud*sdiag0[sInd[jInd][jy[0]]][1];
      dnx[2] = sdiag[sInd[jInd][jy[0]]][2] - Lmud*sdiag0[sInd[jInd][jy[0]]][2];
      locX[0] = rRot[0][0]*dnx[0]+rRot[0][1]*dnx[1]+rRot[0][2]*dnx[2];
      locX[1] = rRot[1][0]*dnx[0]+rRot[1][1]*dnx[1]+rRot[1][2]*dnx[2];
      locX[2] = rRot[2][0]*dnx[0]+rRot[2][1]*dnx[1]+rRot[2][2]*dnx[2];
      dctroid[3*js1] = dctroid[3*js] + locX[0];
      dctroid[3*js1+1] = dctroid[3*js+1] + locX[1];
      dctroid[3*js1+2] = dctroid[3*js+2] + locX[2];
      dtinc[0]+=dtmin;
  }
}

__device__ void loadRotMat(double omega, double *ax, double  rRot[][3])
{
  // loads the rotation matrix from (omega,ax), note that
  // ax is a 3x1  and rRot is a 3x3 static arrays
  rRot[0][0] = cos(omega) + pow(ax[0],2.0)*(1-cos(omega));
  rRot[0][1] = ax[0]*ax[1]*(1-cos(omega)) - ax[2]*sin(omega);
  rRot[0][2] = ax[0]*ax[2]*(1-cos(omega)) + ax[1]*sin(omega);
  rRot[1][0] = ax[0]*ax[1]*(1-cos(omega)) + ax[2]*sin(omega);
  rRot[1][1] = cos(omega) + pow(ax[1],2.0)*(1-cos(omega));
  rRot[1][2] = ax[1]*ax[2]*(1-cos(omega)) - ax[0]*sin(omega);
  rRot[2][0] = ax[2]*ax[0]*(1-cos(omega)) - ax[1]*sin(omega);
  rRot[2][1] = ax[2]*ax[1]*(1-cos(omega)) + ax[0]*sin(omega);
  rRot[2][2] = cos(omega) + pow(ax[2],2.0)*(1-cos(omega));
} 

__device__ void projectPointLine(double *A, double *x0, double *x1, double *xproj)
{
  // computes the coordinate of projecting a point A on line connecting x0 and x1 = xproj
  // note that A,x0,x1,xproj are all length 3 arrays
  double n[3],snorm,d,t;
  snorm = pow(pow(x1[0]-x0[0],2)+pow(x1[1]-x0[1],2)+pow(x1[2]-x0[2],2),.5);
  n[0] = (x1[0]-x0[0])/snorm;
  n[1] = (x1[1]-x0[1])/snorm;
  n[2] = (x1[2]-x0[2])/snorm;
  d = -(n[0]*A[0]+n[1]*A[1]+n[2]*A[2]);
  t = (-d - n[0]*x0[0] - n[1]*x0[1] - n[2]*x0[2])/
    (n[0]*(x1[0]-x0[0])+n[1]*(x1[1]-x0[1])+n[2]*(x1[2]-x0[2]));
  xproj[0] = x0[0] + (x1[0]-x0[0])*t;
  xproj[1] = x0[1] + (x1[1]-x0[1])*t;
  xproj[2] = x0[2] + (x1[2]-x0[2])*t;
} //end projectPointLine...

__device__  void loadS(double S[][3],int sInd[][3])
{
  // this is for decentered octahedron method:
  // S is 6 corners of octahedron in local coor and sInd gives the 3 corner
  // indices for a given octant
  S[0][0]=1;  S[0][1]=0;  S[0][2]=0;
  S[1][0]=0;  S[1][1]=1;  S[1][2]=0;
  S[2][0]=0;  S[2][1]=0;  S[2][2]=1;
  S[3][0]=-1;  S[3][1]=0;  S[3][2]=0;
  S[4][0]=0;  S[4][1]=-1;  S[4][2]=0;
  S[5][0]=0;  S[5][1]=0;  S[5][2]=-1;
  sInd[0][0]=0;  sInd[0][1]=1;  sInd[0][2]=2;
  sInd[1][0]=1;  sInd[1][1]=2;  sInd[1][2]=3;
  sInd[2][0]=0;  sInd[2][1]=2;  sInd[2][2]=4;
  sInd[3][0]=2;  sInd[3][1]=3;  sInd[3][2]=4;
  sInd[4][0]=0;  sInd[4][1]=1;  sInd[4][2]=5;
  sInd[5][0]=1;  sInd[5][1]=3;  sInd[5][2]=5;
  sInd[6][0]=0;  sInd[6][1]=4;  sInd[6][2]=5;
  sInd[7][0]=3;  sInd[7][1]=4;  sInd[7][2]=5;
}// end inline void loadS

__global__ void reduceGlobalArray(int *ig, int n,int isw)
{
  int js,tid=threadIdx.x+blockDim.x*blockIdx.x,stride=blockDim.x*gridDim.x,s,
    nthread=blockDim.x,tidL=threadIdx.x,bidL=blockIdx.x;
  extern __shared__ volatile int sh[];
  if (isw==0){
    // isw=0 is for summation
    js=tid;
    sh[tidL]=0;
    while (js<n){
      sh[tidL]+=ig[js];
      js+=stride;
    }
    __syncthreads();
    s=nthread;
    while (s>=128){
      if (nthread >= s){
	if (tidL<s/2){sh[tidL] += sh[tidL+s/2];}
	__syncthreads();
      }
      s/=2;
    }
    if (tidL<32) {
      if (nthread >=64) {sh[tidL] += sh[tidL+32];}
      if (nthread >=32) {sh[tidL] += sh[tidL+16];}
      if (nthread >=16) {sh[tidL] += sh[tidL+8];}
      if (nthread >= 8) {sh[tidL] += sh[tidL+4];}
      if (nthread >= 4) {sh[tidL] += sh[tidL+2];}
      if (nthread >= 2) {sh[tidL] += sh[tidL+1];}
    }
    if (tidL==0) {
      ig[bidL] = sh[tidL];
    }
  }
}

void resizeGlobalArray(double **y, int &n0, int &n1)
{
  double *d_ctmp;
  HandleError(hipMallocManaged((void**)&d_ctmp,n0*sizeof(double)));
  int nThreads=512;
  int nBlocks=n0/nThreads;
  copyGlobal<<<nBlocks,nThreads>>>(d_ctmp,*y, n0);
  hipFree(*y);
  HandleError(hipMallocManaged((void**)y,n1*sizeof(double)));
  copyGlobal<<<nBlocks,nThreads>>>(*y,d_ctmp, n0);
  hipFree(d_ctmp);
}

void resizeArray(double **y, int &n)
{
  double *tmp;
  tmp=(double*)malloc(n*sizeof(double));
  free(*y);
  *y=tmp;
  tmp=NULL;
  free(tmp);

}

// constructor
VoxelsCA::VoxelsCA(Grid &g)
{ 
  int Ntot = g.nX[0]*g.nX[1]*g.nX[2];
  gID = (int*)malloc(Ntot*sizeof(int));
  vState = (int*)malloc(Ntot*sizeof(int));
  extents = (double*)malloc(Ntot*sizeof(double));
  centroidOct = (double*)malloc(3*Ntot*sizeof(double));
  memset(gID,0,Ntot*sizeof(int));
  memset(vState,0,Ntot*sizeof(int));
  memset(extents,0,Ntot*sizeof(double));
  memset(centroidOct,0,3*Ntot*sizeof(double));
  seed0= 2132512;
  seed1=2912351;
  genlayer.seed(seed1);
} // end constructor
void VoxelsCA::AddLayer1Macro(VoxelsCA *d_vx,Grid &g,Grid *d_g,double **d_cthptr,
		    double *d_troids, int *d_gid, int *d_vst,int &nbuf2)
{
  int npg, nThreads,nBlocks,nbuf1, *d_itmp,Ntot;
  double *d_Sites;
  Ntot=g.nX[0]*g.nX[1]*g.nX[2];
  getNumPowderGrains(g,npg);
  HandleError(hipMallocManaged((void**)&d_Sites,npg*3*sizeof(double)));
  HandleError(hipMallocManaged((void**)&d_itmp,npg*sizeof(int)));
  // below is buffer for size of cTheta to account for nucleation: 100*expected # of new grains in 3 layers
  nbuf2 = 4*(nGrain+ npg + int(ceil(g.nX[0]*g.nX[1]*(g.layerT/g.dX[2])*3*g.rNmax*pow(g.dX[0]*1e6,3.)*100)));
  nbuf1=4*(nGrain);
  resizeGlobalArray(d_cthptr,nbuf1,nbuf2);
  hipDeviceSynchronize();
  nThreads=512;
  nBlocks = npg/nThreads;
  getSites<<<nBlocks,nThreads>>>(d_g,d_vx,d_Sites,npg);
  nBlocks=(g.nZlayer*g.nX[0]*g.nX[1])/nThreads;
  addLayer1Part1<<<nBlocks,nThreads>>>(d_g,d_vx,d_Sites,d_troids,d_gid,d_vst,d_itmp,npg);
  HandleError( hipPeekAtLastError() );
  nBlocks=1;
  nThreads=256;
  addLayer1Part2<<<1,nThreads>>>(d_g,d_vx,*d_cthptr,d_gid,d_itmp,npg);
  HandleError( hipPeekAtLastError() );
  nThreads=512;
  nBlocks=Ntot/nThreads;
  addLayer1Part3<<<nBlocks,nThreads>>>(d_g,d_gid,d_vst);
  HandleError( hipPeekAtLastError() );
  HandleError(hipMemcpy(&(nGrain), &(d_vx->nGrain), sizeof(int), hipMemcpyDeviceToHost));
  hipFree(d_Sites);
  hipFree(d_itmp);
}
void VoxelsCA::CleanLayerMacro(VoxelsCA *dvx,int *dgid,double **dcthetaptr, int &nTot)
{
  int nThreads,nBlocks, *dgidtmp, *dgvflg;
  double *dcthtmp;
  HandleError(hipMalloc((void**)&dgidtmp,nGrain*sizeof(int)));
  HandleError(hipMalloc((void**)&dgvflg,(nGrain+1)*sizeof(int)));
  HandleError(hipMemset(dgidtmp,0,nGrain*sizeof(int)));
  HandleError(hipMemset(dgvflg,0,(nGrain+1)*sizeof(int)));
  HandleError(hipMalloc((void**)&dcthtmp,4*(nGrain)*sizeof(double)));
  nThreads=512;
  nBlocks=nGrain/nThreads+1;
  cleanLayerPart1<<<nBlocks,nThreads>>>(dvx,dgid,dgvflg,dgidtmp,nTot);
  cleanLayerPart2<<<nBlocks,nThreads>>>(dvx,dgvflg,dgidtmp);
  cleanLayerPart3<<<nBlocks,nThreads>>>(dvx,dgid,dgvflg,dgidtmp,
					dcthtmp, *dcthetaptr,nTot);
  cleanLayerPart4<<<nBlocks,nThreads>>>(dvx,dgvflg);  
  hipFree(*dcthetaptr);
  HandleError(hipMemcpy(&(nGrain), &(dvx->nGrain), sizeof(int), hipMemcpyDeviceToHost));
  HandleError(hipMalloc((void**)dcthetaptr,4*(nGrain)*sizeof(double)));
  copyGlobal<<<nBlocks,nThreads>>>(*dcthetaptr,dcthtmp, 4*nGrain);
  hipFree(dgidtmp);
  hipFree(dgvflg);
  hipFree(dcthtmp);
}
void VoxelsCA::ConvertSolid1Macro(Grid *dg,int *vstate,double *dextents,const int &iswitch,int nThreads,const int &ntot)
{
    if (iswitch==0){
    // converts 2 to 3 if all neighbors >=2 (ie mushy to solid)
    // can only use 1 block
    convertSolid1Part1<<<1,nThreads,nThreads*sizeof(int)>>>(dg,vstate,dextents,ntot);
  } else {
    // converts 3 to 2 if any neighbor is 1 (ie solid to mushy)
    // can only use 1 block
    convertSolid1Part2<<<1,nThreads>>>(dg,vstate,ntot);
  }
}
void VoxelsCA::SetLiquid3Macro(Grid *dg,int *dgid, int *dvstate,double *dtempval, double *dexts,int &nThreads, int &nBlocks)
{
  setLiquid3<<<nBlocks,nThreads>>>(dg,dgid,dvstate,dtempval,dexts);
}
void VoxelsCA::UpdateVoxelsMacro(Grid *dg, Grid &gg,VoxelsCA *dvox,int *dgid, int *dvstate,double *dtempval,double *dexts, 
			    double *troids, double *dctheta,int &nThreads, int &nBlocks, int &ntot)
{
  nThreads=1024;
  nBlocks=ntot/nThreads+1;
  SetLiquid3Macro(dg,dgid,dvstate,dtempval,dexts,nThreads,nBlocks);
  int isw=1, *dvs2c, *djindG,*dj1indG,n1;
  ConvertSolid1Macro(dg,dvstate,dexts,isw,nThreads,ntot);
  // calculate # of mushy voxels (vstate=2)
  bool isf=1,*d_isf;
  double *dtinc;
  float *ddtminG;
  nBlocks= ntot/nThreads+1;
  n1=nBlocks;
  HandleError(hipMalloc((void**)&dtinc,sizeof(double)));
  HandleError(hipMalloc((void**)&djindG,n1*sizeof(int)));
  HandleError(hipMalloc((void**)&dvs2c,2*sizeof(int)));
  HandleError(hipMalloc((void**)&d_isf,sizeof(bool)));
  updateVoxelsPart1<<<nBlocks,nThreads,nThreads*sizeof(int)>>>(dvstate,djindG,ntot);
  HandleError( hipPeekAtLastError() );
  while (nBlocks>3){
    n1=nBlocks;
    nBlocks/=2;
    reduceGlobalArray<<<nBlocks,nThreads,nThreads*sizeof(int)>>>(djindG,n1,0);
  }
  n1=nBlocks;
  reduceGlobalArray<<<1,nThreads,nThreads*sizeof(int)>>>(djindG,n1,0);
  // initialize while loop
  updateVoxelsPart2<<<1,1>>>(djindG,dvs2c,d_isf,dtinc);
  HandleError( hipPeekAtLastError() );
  HandleError(hipFree(djindG));
  nThreads=512;
  nBlocks=ntot/nThreads+1;
  HandleError(hipMalloc((void**)&ddtminG,nBlocks*sizeof(float)));
  HandleError(hipMalloc((void**)&djindG,nBlocks*sizeof(int)));
  HandleError(hipMalloc((void**)&dj1indG,nBlocks*sizeof(int)));
  double rX=gg.rNmax*pow(gg.dX[0]*1e6,3.);
  std::default_random_engine g1(30*gg.tInd+seed1);
  std::uniform_real_distribution<double> xrand1(0.0,1.0);
  while (isf){
    nBlocks=ntot/nThreads+1;  
    updateVoxelsPart3<<<nBlocks,nThreads,2*nThreads*sizeof(int)+nThreads*sizeof(float)>>>
      (dg,dgid,dvstate,dctheta,dtempval,troids,dexts,dtinc,dvs2c,d_isf,ddtminG,djindG,dj1indG,ntot);
    HandleError(hipMemcpy(&isf,d_isf,sizeof(bool),hipMemcpyDeviceToHost));
    HandleError( hipPeekAtLastError() );
    while (nBlocks>3){
      n1=nBlocks;
      nBlocks/=2;
      reduceVoxelCapture<<<nBlocks,nThreads,2*nThreads*sizeof(int)+nThreads*sizeof(float)>>>
	(ddtminG,djindG,dj1indG,n1);
    }
    n1=nBlocks;
    reduceVoxelCapture<<<1,nThreads,2*nThreads*sizeof(int)+nThreads*sizeof(float)>>>
      (ddtminG,djindG,dj1indG,n1);
    // index 0 is winning voxel. 
    if (xrand1(g1)<rX){
      // nucleation occurs in voxel
      updateVoxelsPart4<<<1,1>>>(dg,dvox,dgid,dvstate,dctheta,troids,dexts,dvs2c,
				 ddtminG,dj1indG);
      HandleError( hipPeekAtLastError() );
    } else {
      nBlocks=ntot/nThreads+1;
      updateVoxelsPart5<<<nBlocks,nThreads>>>(dg,dvstate,dtempval,dexts,ddtminG,ntot);
      updateVoxelsPart6<<<1,1>>>(dg,dvox,dgid,dvstate,dctheta,troids,dexts,dvs2c,dtinc,
				 ddtminG,djindG,dj1indG);
      HandleError( hipPeekAtLastError() );
      }
  } // while (isf)
  HandleError(hipFree(ddtminG));
  HandleError(hipFree(djindG));
  HandleError(hipFree(dj1indG));
}
void VoxelsCA::WriteToHDF1(const std::string &filename, const Grid &g, const double *tempcurr)
{
  // writes gID, vState, cTheta per voxel
  int Ntot = g.nX[0]*g.nX[1]*g.nX[2],icase;
  std::string hdf5Filename = filename + ".h5";
  std::vector< float> TempOut(Ntot,0),IPFmapBD(3*Ntot,0), IPFmapx(3*Ntot,0), IPFmapy(3*Ntot,0),cth(4*nGrain,0);
  double vBD[3]={0.0,0.0,1.0},omega,ax[3],vCD[3],rRot[3][3],
    vX[3]={1.0,0.0,0.0},vY[3]={0.0,1.0,0.0},xp,yp,x0,y0,m,a,b,c,H,S,V,sMax,ff,p,q,t;
  std::vector<std::vector<double>> triPts(2,std::vector<double>(3,0));
  triPts[0][0]=0.0;
  triPts[0][1]=2./pow(2,.5)/(1.+1./pow(2,.5));
  triPts[0][2]=2./pow(3,.5)/(1.+1./pow(3,.5));
  triPts[1][0]=0.0;
  triPts[1][1]=0.0;
  triPts[1][2]=2./pow(3,.5)/(1.+1./pow(3,.5));
  m=tan(1./2.*atan2(triPts[1][2],triPts[0][2]));
  a=pow(pow(triPts[1][2]-triPts[1][1],2.)+pow(triPts[0][2]-triPts[0][1],2.),.5);
  b=pow(pow(triPts[0][1],2.)+pow(triPts[1][1],2.) ,0.5);
  c=pow(pow(triPts[0][2],2.)+pow(triPts[1][2],2.),0.5);
  y0=1./2.*pow((b+c-a)*(c+a-b)*(a+b-c)/(a+b+c),.5);
  x0=y0/m;
  sMax=pow(pow(x0,2.)+pow(y0,2.),.5);
  for (int j=0;j<Ntot;++j){
    TempOut[j] = tempcurr[j];
    if (gID[j]<1){
      IPFmapBD[3*j] = 0.0;
      IPFmapBD[3*j+1] = 0.0;
      IPFmapBD[3*j+2] = 0.0;
      IPFmapx[3*j] = 0.0;
      IPFmapx[3*j+1] = 0.0;
      IPFmapx[3*j+2] = 0.0;
      IPFmapy[3*j] = 0.0;
      IPFmapy[3*j+1] = 0.0;
      IPFmapy[3*j+2] = 0.0;
    } else {
      omega = cTheta[4*(gID[j]-1)];
      ax[0]= cTheta[4*(gID[j]-1)+1];
      ax[1]= cTheta[4*(gID[j]-1)+2];
      ax[2]= cTheta[4*(gID[j]-1)+3];
      // matrix is local->global; need to multiply by transpose for global->local            
      rRot[0][0] = cos(omega) + pow(ax[0],2.0)*(1-cos(omega));
      rRot[0][1] = ax[0]*ax[1]*(1-cos(omega)) - ax[2]*sin(omega);
      rRot[0][2] = ax[0]*ax[2]*(1-cos(omega)) + ax[1]*sin(omega);
      rRot[1][0] = ax[0]*ax[1]*(1-cos(omega)) + ax[2]*sin(omega);
      rRot[1][1] = cos(omega) + pow(ax[1],2.0)*(1-cos(omega));
      rRot[1][2] = ax[1]*ax[2]*(1-cos(omega)) - ax[0]*sin(omega);
      rRot[2][0] = ax[2]*ax[0]*(1-cos(omega)) - ax[1]*sin(omega);
      rRot[2][1] = ax[2]*ax[1]*(1-cos(omega)) + ax[0]*sin(omega);
      rRot[2][2] = cos(omega) + pow(ax[2],2.0)*(1-cos(omega));
      vCD[0] = std::fabs(rRot[0][0]*vBD[0]+rRot[1][0]*vBD[1]+rRot[2][0]*vBD[2]);
      vCD[1] = std::fabs(rRot[0][1]*vBD[0]+rRot[1][1]*vBD[1]+rRot[2][1]*vBD[2]);
      vCD[2] = std::fabs(rRot[0][2]*vBD[0]+rRot[1][2]*vBD[1]+rRot[2][2]*vBD[2]);
      std::sort(vCD,vCD+3);
      std::swap(vCD[0],vCD[1]);
      vCD[2] = std::min(vCD[2],1.0);
      xp = 2.*vCD[0]/(1.+vCD[2]);
      yp = 2.*vCD[1]/(1.+vCD[2]);
      H=atan( (yp-y0)/(xp-x0))*180./M_PI;
      xp < x0 ? H+=180: H;
      H = H+240-atan((triPts[1][2]-y0)/(triPts[0][2]-x0))*180/M_PI;
      V=1.;
      S=pow(pow(xp-x0,2.)+pow(yp-y0,2.),.5);
      S=S/sMax*0.8 + 0.2;
      H>=360.0 ? H=0.0 : H;
      icase = floor(H/60.0);
      ff= H/6.0 - icase;
      p=V*(1.-S);
      q=V*(1.-S*ff);
      t=V*(1.-(S*(1.-ff)));
      if (S<=0.0){
	IPFmapBD[3*j]=0.0;
	IPFmapBD[3*j+1]=0.0;
	IPFmapBD[3*j+2]=0.0;
      } else {
	if (icase==0){
	  IPFmapBD[3*j]=V;
	  IPFmapBD[3*j+1]=t;
	  IPFmapBD[3*j+2]=p;
	}
	if (icase==1){
	  IPFmapBD[3*j]=q;
	  IPFmapBD[3*j+1]=V;
	  IPFmapBD[3*j+2]=p;
	}
	if (icase==2){
	  IPFmapBD[3*j]=p;
	  IPFmapBD[3*j+1]=V;
	  IPFmapBD[3*j+2]=t;
	}
	if (icase==3){
	  IPFmapBD[3*j]=p;
	  IPFmapBD[3*j+1]=q;
	  IPFmapBD[3*j+2]=V;
	}
	if (icase==4){
	  IPFmapBD[3*j]=t;
	  IPFmapBD[3*j+1]=p;
	  IPFmapBD[3*j+2]=V;
	}
	if (icase==5){
	  IPFmapBD[3*j]=V;
	  IPFmapBD[3*j+1]=p;
	  IPFmapBD[3*j+2]=q;
	}
      }
      // x dir
      vCD[0] = std::fabs(rRot[0][0]*vX[0]+rRot[1][0]*vX[1]+rRot[2][0]*vX[2]);
      vCD[1] = std::fabs(rRot[0][1]*vX[0]+rRot[1][1]*vX[1]+rRot[2][1]*vX[2]);
      vCD[2] = std::fabs(rRot[0][2]*vX[0]+rRot[1][2]*vX[1]+rRot[2][2]*vX[2]);
      std::sort(vCD,vCD+3);
      std::swap(vCD[0],vCD[1]);
      vCD[2]=std::min(vCD[2],1.0);
      xp = 2.*vCD[0]/(1.+vCD[2]);
      yp = 2.*vCD[1]/(1.+vCD[2]);
      H=atan( (yp-y0)/(xp-x0))*180./M_PI;
      xp < x0 ? H+=180: H;
      H = H+240-atan((triPts[1][2]-y0)/(triPts[0][2]-x0))*180/M_PI;
      V=1.;
      S=pow(pow(xp-x0,2.)+pow(yp-y0,2.),.5);
      S=S/sMax*0.8 + 0.2;
      H>=360.0 ? H=0.0 : H;
      icase = floor(H/60.0);
      ff= H/6.0 - icase;
      p=V*(1.-S);
      q=V*(1.-S*ff);
      t=V*(1.-(S*(1.-ff)));
      if (S<=0.0){
	IPFmapx[3*j]=0.0;
	IPFmapx[3*j+1]=0.0;
	IPFmapx[3*j+2]=0.0;
      } else {
	if (icase==0){
	  IPFmapx[3*j]=V;
	  IPFmapx[3*j+1]=t;
	  IPFmapx[3*j+2]=p;
	}
	if (icase==1){
	  IPFmapx[3*j]=q;
	  IPFmapx[3*j+1]=V;
	  IPFmapx[3*j+2]=p;
	}
	if (icase==2){
	  IPFmapx[3*j]=p;
	  IPFmapx[3*j+1]=V;
	  IPFmapx[3*j+2]=t;
	}
	if (icase==3){
	  IPFmapx[3*j]=p;
	  IPFmapx[3*j+1]=q;
	  IPFmapx[3*j+2]=V;
	}
	if (icase==4){
	  IPFmapx[3*j]=t;
	  IPFmapx[3*j+1]=p;
	  IPFmapx[3*j+2]=V;
	}
	if (icase==5){
	  IPFmapx[3*j]=V;
	  IPFmapx[3*j+1]=p;
	  IPFmapx[3*j+2]=q;
	}
      }
      // y dir 
      vCD[0] = std::fabs(rRot[0][0]*vY[0]+rRot[1][0]*vY[1]+rRot[2][0]*vY[2]);
      vCD[1] = std::fabs(rRot[0][1]*vY[0]+rRot[1][1]*vY[1]+rRot[2][1]*vY[2]);
      vCD[2] = std::fabs(rRot[0][2]*vY[0]+rRot[1][2]*vY[1]+rRot[2][2]*vY[2]);
      std::sort(vCD,vCD+3);
      std::swap(vCD[0],vCD[1]);
      vCD[2]=std::min(vCD[2],1.0);
      xp = 2.*vCD[0]/(1.+vCD[2]);
      yp = 2.*vCD[1]/(1.+vCD[2]);
      H = H+240-atan((triPts[1][2]-y0)/(triPts[0][2]-x0))*180/M_PI;
      V=1.;
      S=pow(pow(xp-x0,2.)+pow(yp-y0,2.),.5);
      S=S/sMax*0.8 + 0.2;
      H>=360.0 ? H=0.0 : H;
      icase = floor(H/60.0);
      ff= H/6.0 - icase;
      p=V*(1.-S);
      q=V*(1.-S*ff);
      t=V*(1.-(S*(1.-ff)));
      if (S<=0.0){
	IPFmapy[3*j]=0.0;
	IPFmapy[3*j+1]=0.0;
	IPFmapy[3*j+2]=0.0;
      } else {
	if (icase==0){
	  IPFmapy[3*j]=V;
	  IPFmapy[3*j+1]=t;
	  IPFmapy[3*j+2]=p;
	}
	if (icase==1){
	  IPFmapy[3*j]=q;
	  IPFmapy[3*j+1]=V;
	  IPFmapy[3*j+2]=p;
	}
	if (icase==2){
	  IPFmapy[3*j]=p;
	  IPFmapy[3*j+1]=V;
	  IPFmapy[3*j+2]=t;
	}
	if (icase==3){
	  IPFmapy[3*j]=p;
	  IPFmapy[3*j+1]=q;
	  IPFmapy[3*j+2]=V;
	}
	if (icase==4){
	  IPFmapy[3*j]=t;
	  IPFmapy[3*j+1]=p;
	  IPFmapy[3*j+2]=V;
	}
	if (icase==5){
	  IPFmapy[3*j]=V;
	  IPFmapy[3*j+1]=p;
	  IPFmapy[3*j+2]=q;
	}
      } // if (S<0.0...
    } //     if (gID[j]<1){
  } // for (int j..
  for (int j=0;j<4*nGrain;++j){cth[j]=cTheta[j];}
  unsigned int nVoxT, js,jc,ncth;
  nVoxT = g.nX[0]*g.nX[1]*g.nX[2];
  ncth=4*nGrain;
  js = 0;
  jc = nVoxT;
  std::vector<float> dX(g.dX,g.dX+3);
  adios2::ADIOS adios;
  adios2::IO hdf5IO = adios.DeclareIO("HDFFileIO");
  hdf5IO.SetEngine("HDF5");
  // global array : name, { shape (total) }, { start (local) }, { count (local) }  all are constant dimensions
  adios2::Variable<int> dimsa = hdf5IO.DefineVariable<int>(
	      "dims", {3}, {0}, {3});
  adios2::Variable<float> dxa = hdf5IO.DefineVariable<float>(
	      "VoxelDX", {3}, {0}, {3});
  adios2::Variable<int> gida = hdf5IO.DefineVariable<int>(
	      "gID", {nVoxT}, {js}, {jc});
  adios2::Variable<int> vStatea = hdf5IO.DefineVariable<int>(
	      "vState", {nVoxT}, {js}, {jc});
  adios2::Variable<float> TempOuta = hdf5IO.DefineVariable<float>(
	      "Temperature", {nVoxT}, {js}, {jc});
  adios2::Variable<float> IPFmapBDa = hdf5IO.DefineVariable<float>(
	      "IPFz", {3*nVoxT}, {3*js}, {3*jc});
  adios2::Variable<float> IPFmapxa = hdf5IO.DefineVariable<float>(
	      "IPFx", {3*nVoxT}, {3*js}, {3*jc});
  adios2::Variable<float> IPFmapya = hdf5IO.DefineVariable<float>(
	      "IPFy", {3*nVoxT}, {3*js}, {3*jc});
  adios2::Variable<float> angAx = hdf5IO.DefineVariable<float>(
	      "angleAxis", {ncth}, {0}, {ncth});
  adios2::Engine hdf5Writer =
      hdf5IO.Open(hdf5Filename, adios2::Mode::Write);
  hdf5Writer.Put<int>(dimsa, g.nX);
  hdf5Writer.Put<float>(dxa, dX.data());
  hdf5Writer.Put<int>(gida, gID);
  hdf5Writer.Put<int>(vStatea, vState);
  hdf5Writer.Put<float>(TempOuta, TempOut.data());
  hdf5Writer.Put<float>(IPFmapBDa, IPFmapBD.data());
  hdf5Writer.Put<float>(IPFmapxa, IPFmapx.data());
  hdf5Writer.Put<float>(IPFmapya, IPFmapy.data());
  hdf5Writer.Put<float>(angAx, cth.data());
  hdf5Writer.Close();
} // end WriteToHDF1
